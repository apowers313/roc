
#include <hip/hip_runtime.h>
__global__ void k1() {
  printf("kernel 1 starting...\n");
  hipEvent_t e;
  hipEventCreateWithFlags(&e, hipEventDisableTiming);
  printf("kernel 1 done.\n");
}

__global__ void k2() {
  printf("kernel 2 starting...\n");
  printf("kernel 2 done.\n");
}
